#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void cuda_hello(void)
{
  printf("Hello, world!");
}

int main(void)
{
  printf("Calling cuda_hello...\n");

  // call the CUDA kernel from the GPU
  cuda_hello<<<1,1>>>();

  // wait for the kernel to finish
  hipDeviceSynchronize();

  printf("...device synchronized.\n");

  return 0;
}
