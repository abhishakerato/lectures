#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

/*
  print_array

  Helper function for printing integer arrays of specified length.
*/
void print_array(int* array, size_t length)
{
  printf("[");
  for (size_t i=0; i<length; ++i)
    printf(" %d,", array[i]);
  printf("]\n");
}

/*
  vec_add

  Kernel (device-side) function for computing the sum of two arrays.
 */
__global__ void vec_add(int* out, int* v, int* w)
{
  size_t index = threadIdx.x;
  out[index] = v[index] + w[index];
}



int main(void)
{
  printf("Allocating host-side arrays...");
  size_t N = 16;
  size_t size = N*sizeof(int);
  int* v = (int*) malloc(size);
  int* w = (int*) malloc(size);
  int* out = (int*) malloc(size);

  for (size_t i=0; i<N; ++i)
    {
      v[i] = i;
      w[i] = N-i;
    }
  printf("...done.\n");


  printf("Allocating device-side arrays...");
  int* dev_v;
  int* dev_w;
  int* dev_out;
  hipMalloc((void**) &dev_v, size);
  hipMalloc((void**) &dev_w, size);
  hipMalloc((void**) &dev_out, size);
  printf("...done.\n");


  printf("Copying input host-side arrays to device-side arrays...");
  hipMemcpy(dev_v, v, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_w, w, size, hipMemcpyHostToDevice);
  printf("...done.\n");


  printf("Calling vec_add kernel...");
  vec_add<<<1,N>>>(dev_out, dev_v, dev_w);
  printf("...done.\n");


  // wait for the kernel to finish
  hipError_t cudaerr = hipDeviceSynchronize();
  printf("\t[error message: \"%s\".]\n",
         hipGetErrorString(cudaerr));


  printf("Copying device-side result to host-side array...");
  hipMemcpy(out, dev_out, size, hipMemcpyDeviceToHost);
  printf("...done.\n");


  printf("Printing host-side result...\n\n");
  print_array(out, N);
  printf("\n...done.\n");


  printf("Freeing device-side arrays...");
  hipFree(dev_v);
  hipFree(dev_w);
  hipFree(dev_out);
  printf("...done.\n");


  printf("Freeing host-side arrays...");
  free(v);
  free(w);
  free(out);
  printf("...done.\n");

  return 0;
}
