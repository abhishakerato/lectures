#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void cuda_hello(void)
{
  // print a character buffer from the GPU!
  printf("Hello, world, from thread %d, block %d!\n",
         threadIdx.x, blockIdx.x);
}

int main(void)
{
  printf("Calling cuda_hello...\n");

  // call the CUDA kernel from the GPU
  cuda_hello<<<2,4>>>();

  // wait for the kernel to finish
  hipError_t cudaerr = hipDeviceSynchronize();
  if (cudaerr)
    printf("error message: \"%s\".\n", hipGetErrorString(cudaerr));

  printf("...device synchronized.\n");

  return 0;
}
