#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void cuda_hello(void)
{
  // print a character buffer from the GPU!
  printf("Hello, world!\n");
}

int main(void)
{
  printf("Calling cuda_hello...\n");

  // call the CUDA kernel from the GPU
  cuda_hello<<<1,1>>>();

  // wait for the kernel to finish
  hipDeviceSynchronize();
  printf("...device synchronized.\n");

  return 0;
}
