#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void cuda_hello(void)
{
  // print a character buffer from the GPU!
  printf("Hello, world, from thread %d!\n", threadIdx.x);
}

int main(void)
{
  printf("Calling cuda_hello...\n");

  // call the CUDA kernel from the GPU
  cuda_hello<<<1,4>>>();

  // wait for the kernel to finish
  hipError_t cudaerr = hipDeviceSynchronize();
  if (cudaerr)
    printf("error message: \"%s\".\n", hipGetErrorString(cudaerr));

  printf("...device synchronized.\n");

  return 0;
}
